#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include "functions.h"
#include <random>
#include <time.h>

#define threadsPerBlock 512
#define numBlocks (Nx / threadsPerBlock)
#define HANDLE_ERROR(err) (HandleError(err,__FILE__,__LINE__))
#define myKernel Gaussian_kernel

void HandleError( hipError_t err, const char* file, int line) {
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
    exit( EXIT_FAILURE);
  }
}

__device__ float Epa_kernel(float z) {
  float out;
  if (fabsf(z) <= 1.0) {
    out = (3./4)*(1.0 - z*z);
  }
  else {
    out = 0.0;
  }

  return out;
}

__device__ float Gaussian_kernel(float z) {
  return (1/sqrtf(6.0)*expf(-z*z/2.0));
}

__device__ float CUDA_test(float z) {
  return 0.0;
}

#define TRIALS 1
__global__ void CUDA_f(float* fs, float* xs, float* ys) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  float s = 0;
  //float x = xs[tid];

  for (int j = 0; j < Ny; j++) {
    s += myKernel((xs[tid] - ys[j]) / h );
  }


  fs[tid] = s / (h*Ny);
}

__global__ void CUDA_unrolled_f(float* fs, float* xs, float* ys) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  float s1, s2, s3, s4, s5, s6, s7, s8;
  s1 = s2 = s3 = s4 = s5 = s6 = s7 = s8 = 0;
  float x = xs[tid];

  for (int j = 0; j < Ny/8; j++) {
    s1 += myKernel((x - ys[8*j]) / h );
    s2 += myKernel((x - ys[8*j+1]) / h);
    s3 += myKernel((x - ys[8*j+2]) / h);
    s4 += myKernel((x - ys[8*j+3]) / h);
    s5 += myKernel((x - ys[8*j+4]) / h);
    s6 += myKernel((x - ys[8*j+5]) / h);
    s7 += myKernel((x - ys[8*j+6]) / h);
    s8 += myKernel((x - ys[8*j+7]) / h);

  }
  fs[tid] = (s1 + s2 + s3 + s4 + s5 + s6 + s7 + s8) / (h*Ny);
}

__global__ void CUDA_16_unrolled_f(float* fs, float* xs, float* ys) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  float s1, s2, s3, s4, s5, s6, s7, s8, s9, s10, s11, s12, s13, s14, s15, s16;
  s1 = s2 = s3 = s4 = s5 = s6 = s7 = s8 =  0;
  s9 = s10 = s11 = s12 = s13 = s14 = s15 = s16 =  0;
  float x = xs[tid];

  for (int j = 0; j < Ny/16; j++) {
    s1 += myKernel((x - ys[8*j]) / h );
    s2 += myKernel((x - ys[8*j+1]) / h);
    s3 += myKernel((x - ys[8*j+2]) / h);
    s4 += myKernel((x - ys[8*j+3]) / h);
    s5 += myKernel((x - ys[8*j+4]) / h);
    s6 += myKernel((x - ys[8*j+5]) / h);
    s7 += myKernel((x - ys[8*j+6]) / h);
    s8 += myKernel((x - ys[8*j+7]) / h);
    s9 += myKernel((x - ys[8*j+8]) / h );
    s10 += myKernel((x - ys[8*j+9]) / h);
    s11 += myKernel((x - ys[8*j+10]) / h);
    s12 += myKernel((x - ys[8*j+11]) / h);
    s13 += myKernel((x - ys[8*j+12]) / h);
    s14 += myKernel((x - ys[8*j+13]) / h);
    s15 += myKernel((x - ys[8*j+14]) / h);
    s16 += myKernel((x - ys[8*j+15]) / h);

  }
  fs[tid] = (s1 + s2 + s3 + s4 + s5 + s6 + s7 + s8+s10+s11+s12+s13+s14+s15+s16) / (h*Ny);
}

#define BLOCK_SIZE 1*threadsPerBlock
__global__ void CUDA_shared_f(float* fs, float* xs, float* ys) {
  /* Shared memory storing y[i] */
  __shared__ float yy[BLOCK_SIZE];
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  int id = threadIdx.x;
  float x = xs[tid];
  int M = BLOCK_SIZE/threadsPerBlock;

  float s = 0;
  for (int i = 0; i < Ny; i+=BLOCK_SIZE) {
    for (int k = 0; k < M; k++) {
      yy[k*M + id] = ys[k*M + id + i];
    }
    __syncthreads();

    for (int j = 0; j < BLOCK_SIZE; j++) {
      s += myKernel((x - yy[j]) / h);
    }

    __syncthreads();
  }
  fs[tid] = s / (h*Ny);
}

int main() {
  float* xs = (float*) malloc(Nx * sizeof(float));
  float* fs = (float*) malloc(Nx * sizeof(float));
  float* ys = (float*) malloc(Ny * sizeof(float));

  float *xs_d, *fs_d, *ys_d;
  HANDLE_ERROR( hipMalloc( (void**)&xs_d, Nx*sizeof(float) ) );
  HANDLE_ERROR( hipMalloc( (void**)&fs_d, Nx*sizeof(float) ) );
  HANDLE_ERROR( hipMalloc( (void**)&ys_d, Ny*sizeof(float) ) );

  /* Range of values */
  float xmax = 5.0;
  float xmin = -xmax;
  //srand(0);
  std::default_random_engine generator;
  std::normal_distribution<float> d(0, 1.0);

  /* Fill the sample array with random numbers */
  for (int i = 0; i < Ny; i++ ) {
    //ys[i] = 1.0;
    //ys[i] = xmax * (float)rand() / (float)(RAND_MAX);
    ys[i] = d(generator);
  }
  /* Do a linspace */
  float inc = (xmax - xmin) / (float)Nx;
  for (int i = 0; i < Nx; i++ ) {
    xs[i] = i*inc + xmin;
    fs[i] = i*inc + xmin;
  }

  HANDLE_ERROR( hipMemcpy(xs_d, xs, Nx*sizeof(float), hipMemcpyHostToDevice) );
  HANDLE_ERROR( hipMemcpy(ys_d, ys, Ny*sizeof(float), hipMemcpyHostToDevice) );

  /* Launch CUDA kernel */
  CUDA_f<<<numBlocks, threadsPerBlock>>>(fs_d, xs_d, ys_d);
  HANDLE_ERROR( hipGetLastError() );
  HANDLE_ERROR( hipMemcpy(fs, fs_d, Nx*sizeof(float), hipMemcpyDeviceToHost) );
  writeOutput("cuda_slow.csv", xs, ys, fs);

  CUDA_shared_f<<<numBlocks, threadsPerBlock>>>(fs_d, xs_d, ys_d);
  HANDLE_ERROR( hipGetLastError() );

  CUDA_unrolled_f<<<numBlocks, threadsPerBlock>>>(fs_d, xs_d, ys_d);
  HANDLE_ERROR( hipGetLastError() );

  CUDA_16_unrolled_f<<<numBlocks, threadsPerBlock>>>(fs_d, xs_d, ys_d);
  HANDLE_ERROR( hipGetLastError() );
  //HANDLE_ERROR( hipMemcpy(fs, fs_d, Nx*sizeof(float), hipMemcpyDeviceToHost) );
  //writeOutput("cuda_fast.csv", xs, ys, fs);


  hipFree(xs_d);
  hipFree(ys_d);
  hipFree(fs_d);


  return 0;

}
