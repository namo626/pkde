#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include "functions.h"
#include <random>

#define threadsPerBlock 1024
#define numBlocks (Nx / threadsPerBlock)
#define HANDLE_ERROR(err) (HandleError(err,__FILE__,__LINE__))

void HandleError( hipError_t err, const char* file, int line) {
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
    exit( EXIT_FAILURE);
  }
}

__device__ float CUDA_kernel(float z) {
  float out;
  if (fabsf(z) <= 1.0) {
    out = (3./4)*(1.0 - z*z);
  }
  else {
    out = 0.0;
  }

  return out;
}

__global__ void CUDA_f(float* fs, float* xs, float* ys) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  float s = 0;
  float x = xs[tid];
  float temp;

  for (int j = 0; j < Ny; j++) {
    s += CUDA_kernel((x - ys[j]) / h );
  }

  fs[tid] = s / (h*Ny);
}

int main() {
  float* xs = (float*) malloc(Nx * sizeof(float));
  float* fs = (float*) malloc(Nx * sizeof(float));
  float* ys = (float*) malloc(Ny * sizeof(float));

  float *xs_d, *fs_d, *ys_d;
  HANDLE_ERROR( hipMalloc( (void**)&xs_d, Nx*sizeof(float) ) );
  HANDLE_ERROR( hipMalloc( (void**)&fs_d, Nx*sizeof(float) ) );
  HANDLE_ERROR( hipMalloc( (void**)&ys_d, Ny*sizeof(float) ) );

  /* Range of values */
  float xmax = 5.0;
  float xmin = -xmax;
  //srand(0);
  std::default_random_engine generator;
  std::normal_distribution<float> d(0, 1.0);

  /* Fill the sample array with random numbers */
  for (int i = 0; i < Ny; i++ ) {
    //ys[i] = 1.0;
    //ys[i] = xmax * (float)rand() / (float)(RAND_MAX);
    ys[i] = d(generator);
  }
  /* Do a linspace */
  float inc = (xmax - xmin) / (float)Nx;
  for (int i = 0; i < Nx; i++ ) {
    xs[i] = i*inc + xmin;
    fs[i] = i*inc + xmin;
  }

  HANDLE_ERROR( hipMemcpy(xs_d, xs, Nx*sizeof(float), hipMemcpyHostToDevice) );
  HANDLE_ERROR( hipMemcpy(ys_d, ys, Ny*sizeof(float), hipMemcpyHostToDevice) );

  /* Launch CUDA kernel */
  CUDA_f<<<numBlocks, threadsPerBlock>>>(fs_d, xs_d, ys_d);
  HANDLE_ERROR( hipGetLastError() );

  HANDLE_ERROR( hipMemcpy(fs, fs_d, Nx*sizeof(float), hipMemcpyDeviceToHost) );

  hipFree(xs_d);
  hipFree(ys_d);
  hipFree(fs_d);

  /* write output */
  writeOutput("cuda.csv", xs, ys, fs);

  return 0;

}